#include "hip/hip_runtime.h"

#include "Process_CUDA.h"

#ifdef USE_CUDA
#include <assert.h>
#include <stdio.h>
#include <conio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include ""
#include "hip/device_functions.h"

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))


#define GetRectWidth(x) (x.right - x.left + 1)
#define GetRectHeight(x) (x.bottom - x.top + 1)
#define round(a) ((a>0.0f) ? (int)(a + 0.5f) : (int)(a - 0.5f))
#define round2(a, b) ((a> 0) ? (a + b / 2) : (a - b / 2))

void cudaProcess::AllocCudaMem()
{
	if (m_width == 0 || m_height == 0)
		return;

	int mem_size = m_width*m_height;
	hipMallocManaged((void**)&m_dpbGray, mem_size);
	hipMallocManaged((void**)&m_dpnEdge, mem_size*sizeof(int));
	hipMallocManaged((void**)&m_dpbEdgeSpec, mem_size);
	hipMallocManaged((void**)&m_dpbTemp, mem_size);
}
void cudaProcess::FreeCudaMem()
{
	hipFree(m_dpbGray);
	hipFree(m_dpnEdge);
	hipFree(m_dpbEdgeSpec);
	hipFree(m_dpbTemp);
}


__global__ void
    DetectEdgeKernel(BYTE* pbGray, BYTE* pbEdge, int nWidth, int nHeight)
{

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = i / nWidth;
	int idx = i - idy * nWidth;
	int pos = idy*nWidth + idx;
	int subSum, localSum;
	
	if (idx < 0 || idx >= nWidth || idy < 0 || idy >= nHeight)
		return;
	if (idx <= 3 || idy <= 3 || idy >= nHeight - 3 || idx >= nWidth - 3)
	{
		pbEdge[pos] = 128;//white
		return;
	}
	subSum = localSum = 0;
	for (int y = idy - 3; y <= idy + 3; y++)
		for (int x = idx - 3; x <= idx + 3; x++)
			subSum += pbGray[y*nWidth + x];
	localSum = pbGray[pos] + pbGray[pos - 1] + pbGray[pos + 1] + pbGray[pos-nWidth] + pbGray[pos+nWidth];
	subSum -= localSum;
	subSum /= 44;
	localSum /= 5;
	if (subSum - localSum > 3)
		pbEdge[pos] = 0;
	else if (subSum - localSum < -3)
		pbEdge[pos] = 255;
	else
		pbEdge[pos] = 128;
}

void cudaProcess::setGrayData(BYTE* pbGray)
{
	int mem_size = m_width*m_height;
	hipMemcpy(m_dpbGray, pbGray, mem_size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (mem_size + threadsPerBlock - 1) / threadsPerBlock;
	DetectEdgeKernel << < blocksPerGrid, threadsPerBlock >> >(m_dpbGray, m_dpbEdgeSpec, m_width, m_height);
}

void cudaProcess::getEdgeSpec(BYTE* pbEdgeSpec)
{
	int mem_size = m_width*m_height;
	//int threadsPerBlock = 256;
	//int blocksPerGrid = (mem_size + threadsPerBlock - 1) / threadsPerBlock;
	//DetectEdgeKernel <<< blocksPerGrid, threadsPerBlock >>>(m_dpbGray, m_dpbEdgeSpec, m_width, m_height);
	hipMemcpy(pbEdgeSpec, m_dpbEdgeSpec, mem_size, hipMemcpyDeviceToHost);

}

#endif